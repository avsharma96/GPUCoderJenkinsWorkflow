#include "hip/hip_runtime.h"
//
//  stereoDisparity.cu
//
//  Code generation for function 'stereoDisparity'
//


// Include files
#include "stereoDisparity.h"
#include "MWCudaDimUtility.hpp"
#include "rt_nonfinite.h"

// Function Declarations
static __global__ void stereoDisparity_kernel1(real_T out_disp[145408], real_T
  min_cost[145408]);
static __global__ void stereoDisparity_kernel2(const uint8_T img1[581632], const
  uint8_T img0[581632], const int32_T d, int32_T diff_img[158400]);
static __global__ void stereoDisparity_kernel3(const int32_T diff_img[158400],
  real_T cost_v[153600]);
static __global__ void stereoDisparity_kernel4(const real_T cost_v[153600],
  real_T cost[145408]);
static __global__ void stereoDisparity_kernel5(const int32_T d, const real_T
  cost[145408], real_T out_disp[145408], real_T min_cost[145408]);

// Function Definitions
static __global__ __launch_bounds__(512, 1) void stereoDisparity_kernel1(real_T
  out_disp[145408], real_T min_cost[145408])
{
  uint32_T threadId;
  int32_T temp_cost;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  temp_cost = static_cast<int32_T>(threadId);
  if (temp_cost < 145408) {
    //  Modified Algorithm for Stereo Disparity Block Matching
    //  In this implementation, instead of finding shifted image, indices are
    //  mapped accordingly to save memory and some processing. RGBA column major  
    //  packed data is used as input for compatibility with CUDA intrinsics.
    //  Convolution is performed using separable filters (horizontal and then
    //  vertical).
    //    Copyright 2017-2019 The MathWorks, Inc.
    //  GPU code generation pragma
    //  Stereo Disparity Parameters
    //  |WIN_RAD| is the radius of the window to be operated. |min_disparity| is  
    //  the minimum disparity level the search continues for. |max_disparity| is  
    //  the maximum disparity level the search continues for.
    //  Image Dimensions for Loop Control
    //  The number of channels packed are 4 (RGBA) so as nChannels are 4.
    //  Store the Raw Differences
    //  Store the minimum cost
    //  Store the final disparity
    min_cost[temp_cost] = 9.9999999E+7;
    out_disp[temp_cost] = 0.0;
  }
}

static __global__ __launch_bounds__(512, 1) void stereoDisparity_kernel2(const
  uint8_T img1[581632], const uint8_T img0[581632], const int32_T d, int32_T
  diff_img[158400])
{
  uint32_T threadId;
  int32_T tDiff;
  int32_T colIdx;
  int32_T rowIdx;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  rowIdx = static_cast<int32_T>(threadId % 300U);
  colIdx = static_cast<int32_T>((threadId - static_cast<uint32_T>(rowIdx)) /
    300U);
  if (colIdx < 528) {
    int32_T ind_w2;
    int32_T ind_w1;
    int32_T ind_h;

    //  Row index calculation.
    ind_h = rowIdx - 7;

    //  Column indices calculation for left image.
    ind_w1 = colIdx - 7;

    //  Row indices calculation for right image.
    ind_w2 = (colIdx + d) - 23;

    //  Border clamping for row Indices.
    if (rowIdx - 7 <= 0) {
      ind_h = 1;
    }

    if (ind_h > 284) {
      ind_h = 284;
    }

    //  Border clamping for column indices for left image.
    if (colIdx - 7 <= 0) {
      ind_w1 = 1;
    }

    if (ind_w1 > 512) {
      ind_w1 = 512;
    }

    //  Border clamping for column indices for right image.
    if (ind_w2 <= 0) {
      ind_w2 = 1;
    }

    if (ind_w2 > 512) {
      ind_w2 = 512;
    }

    //  In this step, Sum of absolute Differences is performed
    //  across tour channels.
    tDiff = 0;
    for (int32_T kk = 0; kk < 4; kk++) {
      int32_T temp_cost;
      temp_cost = static_cast<int32_T>(img0[(((ind_h - 1) << 2) + kk) + 1136 *
        (ind_w1 - 1)]) - static_cast<int32_T>(img1[(((ind_h - 1) << 2) + kk) +
        1136 * (ind_w2 - 1)]);
      if (temp_cost < 0) {
        temp_cost = -temp_cost;
      }

      if ((static_cast<int32_T>(tDiff < 0)) && (static_cast<int32_T>(temp_cost <
            MIN_int32_T - tDiff))) {
        tDiff = MIN_int32_T;
      } else if ((static_cast<int32_T>(tDiff > 0)) && (static_cast<int32_T>
                  (temp_cost > MAX_int32_T - tDiff))) {
        tDiff = MAX_int32_T;
      } else {
        tDiff += temp_cost;
      }

      // tDiff = tDiff + abs(int32(img0((ind_h-1)*(nChannels)+chIdx,ind_w1))-int32(img1((ind_h-1)*(nChannels)+chIdx,ind_w2))); 
    }

    //  Store the SAD cost into a matrix.
    diff_img[rowIdx + 300 * colIdx] = tDiff;
  }
}

static __global__ __launch_bounds__(1024, 1) void stereoDisparity_kernel3(const
  int32_T diff_img[158400], real_T cost_v[153600])
{
  int32_T colIdx;
  real_T cv;
  int32_T kk;
  __shared__ int32_T diff_img_shared[1536];
  int32_T baseR;
  int32_T srow;
  int32_T strideRow;
  int32_T scol;
  int32_T strideCol;
  int32_T baseC;
  int32_T x_idx;
  colIdx = mwGetGlobalThreadIndexInYDimension();
  kk = mwGetGlobalThreadIndexInXDimension();
  baseR = kk;
  srow = static_cast<int32_T>(threadIdx.x);
  strideRow = static_cast<int32_T>(blockDim.x);
  scol = static_cast<int32_T>(threadIdx.y);
  strideCol = static_cast<int32_T>(blockDim.y);
  while (srow <= 31) {
    baseC = colIdx;
    for (x_idx = scol; x_idx <= 47; x_idx += strideCol) {
      if ((static_cast<int32_T>((static_cast<int32_T>(baseR >= 0)) && (
             static_cast<int32_T>(baseR < 300)))) && (static_cast<int32_T>((
             static_cast<int32_T>(baseC >= 0)) && (static_cast<int32_T>(baseC <
              528))))) {
        diff_img_shared[srow + 32 * x_idx] = diff_img[300 * baseC + baseR];
      } else {
        diff_img_shared[srow + 32 * x_idx] = 0;
      }

      baseC += strideCol;
    }

    baseR += strideRow;
    srow += strideRow;
  }

  __syncthreads();
  if ((static_cast<int32_T>(colIdx < 512)) && (static_cast<int32_T>(kk < 300)))
  {
    cv = 0.0;
    for (baseR = 0; baseR < 17; baseR++) {
      cv += static_cast<real_T>(diff_img_shared[(static_cast<int32_T>
        (threadIdx.x) + (mwGetGlobalThreadIndexInXDimension() -
                         mwGetGlobalThreadIndexInXDimension())) + 32 * (
        static_cast<int32_T>(threadIdx.y) + baseR)]);
    }

    cost_v[kk + 300 * colIdx] = cv;
  }
}

static __global__ __launch_bounds__(1024, 1) void stereoDisparity_kernel4(const
  real_T cost_v[153600], real_T cost[145408])
{
  int32_T colIdx;
  real_T cv;
  int32_T kk;
  __shared__ real_T cost_v_shared[1536];
  int32_T baseR;
  int32_T srow;
  int32_T strideRow;
  int32_T scol;
  int32_T strideCol;
  int32_T baseC;
  int32_T x_idx;
  colIdx = mwGetGlobalThreadIndexInYDimension();
  kk = mwGetGlobalThreadIndexInXDimension();
  baseR = kk;
  srow = static_cast<int32_T>(threadIdx.x);
  strideRow = static_cast<int32_T>(blockDim.x);
  scol = static_cast<int32_T>(threadIdx.y);
  strideCol = static_cast<int32_T>(blockDim.y);
  while (srow <= 47) {
    baseC = colIdx;
    for (x_idx = scol; x_idx <= 31; x_idx += strideCol) {
      if ((static_cast<int32_T>((static_cast<int32_T>(baseR >= 0)) && (
             static_cast<int32_T>(baseR < 300)))) && (static_cast<int32_T>((
             static_cast<int32_T>(baseC >= 0)) && (static_cast<int32_T>(baseC <
              512))))) {
        cost_v_shared[srow + 48 * x_idx] = cost_v[300 * baseC + baseR];
      } else {
        cost_v_shared[srow + 48 * x_idx] = 0.0;
      }

      baseC += strideCol;
    }

    baseR += strideRow;
    srow += strideRow;
  }

  __syncthreads();
  if ((static_cast<int32_T>(colIdx < 512)) && (static_cast<int32_T>(kk < 284)))
  {
    cv = 0.0;
    for (baseR = 0; baseR < 17; baseR++) {
      cv += cost_v_shared[(static_cast<int32_T>(threadIdx.x) + baseR) + 48 * (
        static_cast<int32_T>(threadIdx.y) + (mwGetGlobalThreadIndexInYDimension()
        - mwGetGlobalThreadIndexInYDimension()))];
    }

    cost[kk + 284 * colIdx] = cv;
  }
}

static __global__ __launch_bounds__(512, 1) void stereoDisparity_kernel5(const
  int32_T d, const real_T cost[145408], real_T out_disp[145408], real_T
  min_cost[145408])
{
  uint32_T threadId;
  int32_T colIdx;
  int32_T kk;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  kk = static_cast<int32_T>(threadId % 284U);
  colIdx = static_cast<int32_T>((threadId - static_cast<uint32_T>(kk)) / 284U);
  if (colIdx < 512) {
    int32_T temp_cost;
    real_T cv;

    //  load the cost
    cv = cost[kk + 284 * colIdx];
    if (cv < 2.147483648E+9) {
      if (cv >= -2.147483648E+9) {
        temp_cost = static_cast<int32_T>(cv);
      } else {
        temp_cost = MIN_int32_T;
      }
    } else if (cv >= 2.147483648E+9) {
      temp_cost = MAX_int32_T;
    } else {
      temp_cost = 0;
    }

    //  Compare against the minimum cost available and store the
    //  disparity value.
    if (min_cost[kk + 284 * colIdx] > static_cast<real_T>(temp_cost)) {
      min_cost[kk + 284 * colIdx] = static_cast<real_T>(temp_cost);
      out_disp[kk + 284 * colIdx] = fabs(static_cast<real_T>(d) + -16.0) + 8.0;
    }
  }
}

void stereoDisparity(const uint8_T img0[581632], const uint8_T img1[581632],
                     real_T out_disp[145408])
{
  real_T (*gpu_out_disp)[145408];
  real_T (*gpu_min_cost)[145408];
  uint8_T (*gpu_img1)[581632];
  uint8_T (*gpu_img0)[581632];
  int32_T (*gpu_diff_img)[158400];
  real_T (*gpu_cost_v)[153600];
  real_T (*gpu_cost)[145408];
  boolean_T img1_dirtyOnCpu;
  boolean_T img0_dirtyOnCpu;
  hipMalloc(&gpu_min_cost, 1163264ULL);
  hipMalloc(&gpu_out_disp, 1163264ULL);
  hipMalloc(&gpu_cost, 1163264ULL);
  hipMalloc(&gpu_cost_v, 1228800ULL);
  hipMalloc(&gpu_diff_img, 633600ULL);
  hipMalloc(&gpu_img0, 581632ULL);
  hipMalloc(&gpu_img1, 581632ULL);
  img1_dirtyOnCpu = true;
  img0_dirtyOnCpu = true;

  //  Modified Algorithm for Stereo Disparity Block Matching
  //  In this implementation, instead of finding shifted image, indices are
  //  mapped accordingly to save memory and some processing. RGBA column major
  //  packed data is used as input for compatibility with CUDA intrinsics.
  //  Convolution is performed using separable filters (horizontal and then
  //  vertical).
  //    Copyright 2017-2019 The MathWorks, Inc.
  //  GPU code generation pragma
  //  Stereo Disparity Parameters
  //  |WIN_RAD| is the radius of the window to be operated. |min_disparity| is
  //  the minimum disparity level the search continues for. |max_disparity| is
  //  the maximum disparity level the search continues for.
  //  Image Dimensions for Loop Control
  //  The number of channels packed are 4 (RGBA) so as nChannels are 4.
  //  Store the Raw Differences
  //  Store the minimum cost
  //  Store the final disparity
  stereoDisparity_kernel1<<<dim3(284U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_out_disp, *gpu_min_cost);

  //  Filters for Aggregating the Differences
  //  |filter_h| is the horizontal filter used in separable convolution.
  //  |filter_v| is the vertical filter used in separable convolution which
  //  operates on the output of the row convolution.
  // if isequal(class(flag), 'int32')
  //     Diff = int32(0);
  // elseif isequal(class(flag),'int16')
  //     Diff = int16(0);
  // else
  //     Diff = int8(0);
  // end
  //  Main Loop that runs for all the disparity levels. This loop is
  //  expected to run on CPU.
  for (int32_T d = 0; d < 17; d++) {
    //  Find the difference matrix for the current disparity level. Expect
    //  this to generate a Kernel function.
    if (img1_dirtyOnCpu) {
      hipMemcpy(gpu_img1, (void *)&img1[0], 581632ULL, hipMemcpyHostToDevice);
      img1_dirtyOnCpu = false;
    }

    if (img0_dirtyOnCpu) {
      hipMemcpy(gpu_img0, (void *)&img0[0], 581632ULL, hipMemcpyHostToDevice);
      img0_dirtyOnCpu = false;
    }

    stereoDisparity_kernel2<<<dim3(310U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (*gpu_img1, *gpu_img0, d, *gpu_diff_img);

    //  Aggregating the differences using separable convolution. Expect this
    //  to generate two kernels using shared memory.The first kernel is the
    //  convolution with the horizontal kernel and second kernel operates on
    //  its output the column wise convolution.
    stereoDisparity_kernel3<<<dim3(10U, 16U, 1U), dim3(32U, 32U, 1U)>>>
      (*gpu_diff_img, *gpu_cost_v);
    stereoDisparity_kernel4<<<dim3(9U, 16U, 1U), dim3(32U, 32U, 1U)>>>
      (*gpu_cost_v, *gpu_cost);

    //  This part updates the min_cost matrix with by comparing the values
    //  with current disparity level.
    stereoDisparity_kernel5<<<dim3(284U, 1U, 1U), dim3(512U, 1U, 1U)>>>(d,
      *gpu_cost, *gpu_out_disp, *gpu_min_cost);
  }

  hipMemcpy(&out_disp[0], gpu_out_disp, 1163264ULL, hipMemcpyDeviceToHost);
  hipFree(*gpu_img1);
  hipFree(*gpu_img0);
  hipFree(*gpu_diff_img);
  hipFree(*gpu_cost_v);
  hipFree(*gpu_cost);
  hipFree(*gpu_out_disp);
  hipFree(*gpu_min_cost);
}

// End of code generation (stereoDisparity.cu)
