//
//  stereoDisparity_initialize.cu
//
//  Code generation for function 'stereoDisparity_initialize'
//


// Include files
#include "stereoDisparity_initialize.h"
#include "_coder_stereoDisparity_mex.h"
#include "rt_nonfinite.h"
#include "stereoDisparity.h"
#include "stereoDisparity_data.h"

// Function Definitions
void stereoDisparity_initialize()
{
  mex_InitInfAndNan();
  mexFunctionCreateRootTLS();
  emlrtClearAllocCountR2012b(emlrtRootTLSGlobal, false, 0U, 0);
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLicenseCheckR2012b(emlrtRootTLSGlobal, "Distrib_Computing_Toolbox", 2);
  emlrtFirstTimeR2012b(emlrtRootTLSGlobal);
  hipGetLastError();
}

// End of code generation (stereoDisparity_initialize.cu)
