//
//  stereoDisparity_terminate.cu
//
//  Code generation for function 'stereoDisparity_terminate'
//


// Include files
#include "stereoDisparity_terminate.h"
#include "_coder_stereoDisparity_mex.h"
#include "rt_nonfinite.h"
#include "stereoDisparity.h"
#include "stereoDisparity_data.h"

// Function Definitions
void stereoDisparity_atexit()
{
  mexFunctionCreateRootTLS();
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
  emlrtExitTimeCleanup(&emlrtContextGlobal);
}

void stereoDisparity_terminate()
{
  hipError_t errCode;
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    emlrtThinCUDAError(false, emlrtRootTLSGlobal);
  }

  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
}

// End of code generation (stereoDisparity_terminate.cu)
