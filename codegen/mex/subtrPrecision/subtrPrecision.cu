#include "hip/hip_runtime.h"
//
//  subtrPrecision.cu
//
//  Code generation for function 'subtrPrecision'
//


// Include files
#include "subtrPrecision.h"
#include "_coder_subtrPrecision_mex.h"
#include "rt_nonfinite.h"
#include <math.h>

// Variable Definitions
emlrtCTX emlrtRootTLSGlobal = NULL;
emlrtContext emlrtContextGlobal = { true,// bFirstTime
  false,                               // bInitialized
  131594U,                             // fVersionInfo
  NULL,                                // fErrorFunction
  "subtrPrecision",                    // fFunctionName
  NULL,                                // fRTCallStack
  false,                               // bDebugMode
  { 3194014219U, 53452778U, 2865749887U, 3590888462U },// fSigWrd
  NULL                                 // fSigMem
};

// Function Declarations
static real_T b_emlrt_marshallIn(const mxArray *u, const emlrtMsgIdentifier
  *parentId);
static real_T c_emlrt_marshallIn(const mxArray *src, const emlrtMsgIdentifier
  *msgId);
static real_T emlrt_marshallIn(const mxArray *a, const char_T *identifier);
static const mxArray *emlrt_marshallOut(const int8_T u);
static real_T rt_roundd_snf(real_T u);

// Function Definitions
static real_T b_emlrt_marshallIn(const mxArray *u, const emlrtMsgIdentifier
  *parentId)
{
  real_T y;
  y = c_emlrt_marshallIn(emlrtAlias(u), parentId);
  emlrtDestroyArray(&u);
  return y;
}

static real_T c_emlrt_marshallIn(const mxArray *src, const emlrtMsgIdentifier
  *msgId)
{
  real_T ret;
  static const int32_T dims = 0;
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src, "double", false, 0U,
    &dims);
  ret = *(real_T *)emlrtMxGetData(src);
  emlrtDestroyArray(&src);
  return ret;
}

static real_T emlrt_marshallIn(const mxArray *a, const char_T *identifier)
{
  real_T y;
  emlrtMsgIdentifier thisId;
  thisId.fIdentifier = const_cast<const char *>(identifier);
  thisId.fParent = NULL;
  thisId.bParentIsCell = false;
  y = b_emlrt_marshallIn(emlrtAlias(a), &thisId);
  emlrtDestroyArray(&a);
  return y;
}

static const mxArray *emlrt_marshallOut(const int8_T u)
{
  const mxArray *y;
  const mxArray *m;
  y = NULL;
  m = emlrtCreateNumericMatrix(1, 1, mxINT8_CLASS, mxREAL);
  *(int8_T *)emlrtMxGetData(m) = u;
  emlrtAssign(&y, m);
  return y;
}

static real_T rt_roundd_snf(real_T u)
{
  real_T y;
  if (fabs(u) < 4.503599627370496E+15) {
    if (u >= 0.5) {
      y = floor(u + 0.5);
    } else if (u > -0.5) {
      y = u * 0.0;
    } else {
      y = ceil(u - 0.5);
    }
  } else {
    y = u;
  }

  return y;
}

int8_T subtrPrecision(real_T a, real_T b)
{
  int8_T out;
  real_T d;
  int8_T i;
  int8_T i1;
  int32_T i2;
  int32_T i3;
  d = rt_roundd_snf(a);
  if (d < 128.0) {
    if (d >= -128.0) {
      i = static_cast<int8_T>(d);
    } else {
      i = MIN_int8_T;
    }
  } else if (d >= 128.0) {
    i = MAX_int8_T;
  } else {
    i = 0;
  }

  d = rt_roundd_snf(b);
  if (d < 128.0) {
    if (d >= -128.0) {
      i1 = static_cast<int8_T>(d);
    } else {
      i1 = MIN_int8_T;
    }
  } else if (d >= 128.0) {
    i1 = MAX_int8_T;
  } else {
    i1 = 0;
  }

  i2 = i - i1;
  if (i2 > 127) {
    i2 = 127;
  } else {
    if (i2 < -128) {
      i2 = -128;
    }
  }

  i3 = -i2;
  if (i3 > 127) {
    i3 = 127;
  }

  if (i2 < 0) {
    out = static_cast<int8_T>(i3);
  } else {
    out = static_cast<int8_T>(i2);
  }

  // works best with int32 precision
  return out;
}

void subtrPrecision_api(const mxArray * const prhs[2], int32_T, const mxArray
  *plhs[1])
{
  real_T a;
  real_T b;

  // Marshall function inputs
  a = emlrt_marshallIn(emlrtAliasP(prhs[0]), "a");
  b = emlrt_marshallIn(emlrtAliasP(prhs[1]), "b");

  // Invoke the target function
  // Marshall function outputs
  plhs[0] = emlrt_marshallOut(subtrPrecision(a, b));
}

void subtrPrecision_atexit()
{
  mexFunctionCreateRootTLS();
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
  emlrtExitTimeCleanup(&emlrtContextGlobal);
}

void subtrPrecision_initialize()
{
  mex_InitInfAndNan();
  mexFunctionCreateRootTLS();
  emlrtClearAllocCountR2012b(emlrtRootTLSGlobal, false, 0U, 0);
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLicenseCheckR2012b(emlrtRootTLSGlobal, "Distrib_Computing_Toolbox", 2);
  emlrtFirstTimeR2012b(emlrtRootTLSGlobal);
  hipGetLastError();
}

void subtrPrecision_terminate()
{
  hipError_t errCode;
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    emlrtThinCUDAError(false, emlrtRootTLSGlobal);
  }

  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
}

// End of code generation (subtrPrecision.cu)
